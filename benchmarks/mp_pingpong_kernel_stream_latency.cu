#include "hip/hip_runtime.h"
/****
 * Copyright (c) 2011-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 ****/

#include <string.h>
#include <stdio.h>
#include <assert.h>
#include <limits.h>
#include <sys/types.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <mpi.h>
#include <gdsync.h>
#include <mp.h>
#include "mp/device.cuh"
#include <vector>

#define NULL_CHECK(ptr)                                 \
do {                                                    \
    if (ptr == NULL) {                                  \
        fprintf(stderr, "[%s:%d] memory allocation failed \n", \
         __FILE__, __LINE__);				\
        exit(-1);                                       \
    }                                                   \
} while (0)

#define CUDA_CHECK(stmt)                                \
do {                                                    \
    hipError_t result = (stmt);                        \
    if (hipSuccess != result) {                        \
        fprintf(stderr, "[%s:%d] cuda failed with %s \n",   \
         __FILE__, __LINE__,hipGetErrorString(result));\
        exit(-1);                                       \
    }                                                   \
    assert(hipSuccess == result);                      \
} while (0)

#define CU_CHECK(stmt)                                 \
do {                                                    \
    hipError_t result = (stmt);                           \
    if (hipSuccess != result) {                        \
        fprintf(stderr, "[%s:%d] cuda failed with %d \n",   \
         __FILE__, __LINE__, result);\
        exit(-1);                                       \
    }                                                   \
    assert(hipSuccess == result);                     \
} while (0)

#define MP_CHECK(stmt)                                  \
do {                                                    \
    int result = (stmt);                                \
    if (0 != result) {                                  \
        fprintf(stderr, "[%s:%d] mp call failed \n",    \
         __FILE__, __LINE__);                           \
        exit(-1);                                       \
    }                                                   \
    assert(0 == result);                                \
} while (0)

int enable_debug_prints = 0;
#define mp_dbg_msg(FMT, ARGS...)  do                                    \
{                                                                       \
    if (enable_debug_prints)  {                                              \
        fprintf(stderr, "[%d] [%d] MP DBG  %s() " FMT, getpid(),  my_rank, __FUNCTION__ , ## ARGS); \
        fflush(stderr);                                                 \
    }                                                                   \
} while(0)

#define MAX_SIZE 256*1024
#define ITER_COUNT_SMALL 1024
#define ITER_COUNT_LARGE 1024

int comm_size, my_rank, peer;
int steps_per_batch = 16, batches_inflight = 4;
int enable_async = 1;

int num_streams = 1;
volatile uint32_t tracking_event = 0;

//calc kernel params
int gpu_num_sm;
static const int over_sub_factor = 2;
int calc_size = 128*1024;
int use_calc_kernel = 0;
uint32_t windex_max;
uint32_t sindex_max;

__device__ int my_rank_d;
__device__ uint32_t windex_max_d;
__device__ uint32_t sindex_max_d;

FILE *cputimes,*streamtimes;
//per-stream state
typedef struct {
    float *in = NULL;
    float *out = NULL;
    void *buf_d;
    mp_request_t *sreq;
    mp_request_t *rreq;
    mp::mlx5::send_desc_t *sdesc;
    mp::mlx5::send_desc_t *sdesc_d;
    mp::mlx5::wait_desc_t *wdesc;
    mp::mlx5::wait_desc_t *wdesc_d;
    mp_reg_t reg; 
    uint32_t sindex;
    uint32_t windex;
    uint32_t *sindex_d;
    uint32_t *windex_d;
    hipGraph_t subgraph;
    hipGraph_t subgraph_comms;
    hipStream_t stream;
} stream_state_t; 

typedef struct {
    int size;
    int batch_index;
} graph_preparation_arg_t;

//global state
stream_state_t *stream_state;
hipStream_t main_stream;
size_t buf_size; 
int capture_graph = 0;
hipGraphNode_t emptyNode; 
hipGraph_t graph, graph_comms;
hipGraph_t subgraph, subgraph_comms;
hipGraphExec_t graphexec, graphexec_comms;

graph_preparation_arg_t *graph_pre_arg;

//timing 
hipEvent_t iter_start_event, iter_stop_event;
hipEvent_t timer_start_event, timer_stop_event;
volatile int *delay_flag;
volatile int *delay_flag_dptr;
__device__ int counter;
__device__ int clockrate;

__global__ void calc_kernel(int n, float c, float *in, float *out)
{
        const uint tid = threadIdx.x;
        const uint bid = blockIdx.x;
        const uint block_size = blockDim.x;
        const uint grid_size = gridDim.x;
        const uint gid = tid + bid*block_size;
        const uint n_threads = block_size*grid_size;
        for (int i=gid; i<n; i += n_threads)
                out[i] = in[i] * c;
}

int gpu_launch_calc_kernel(size_t size, hipStream_t stream)
{
        const int nblocks = over_sub_factor * gpu_num_sm;
        const int nthreads = 32*2;
        int n = size / sizeof(float);
        static float *in = NULL;
        static float *out = NULL;
        if (!in) {
                CUDA_CHECK(hipMalloc((void **)&in, size));
                CUDA_CHECK(hipMalloc((void **)&out, size));

                CUDA_CHECK(hipMemset((void *)in, 1, size));
                CUDA_CHECK(hipMemset((void *)out, 1, size));
        }
        calc_kernel<<<nblocks, nthreads, 0, stream>>>(n, 1.0f, in, out);
        CUDA_CHECK(hipGetLastError());
        return 0;
}

__global__ void poll_kernel(long long int time)
{
    long long int start, stop;
    long long int usec;

    asm volatile("mov.u64  %0, %globaltimer;" : "=l"(start));
    do {
        asm volatile("mov.u64  %0, %globaltimer;" : "=l"(stop));
        assert(stop >= start);
        usec = ((stop-start)/1000);
        counter = usec;
    } while(usec < time);
}

int batch_to_rreq_idx (int batch_idx) { 
     return (batch_idx % (batches_inflight + 1))*steps_per_batch;
}

int batch_to_sreq_idx (int batch_idx) { 
     return (batch_idx % batches_inflight)*steps_per_batch;
}

void post_recv (int size, int batch_index)
{
    int req_idx = batch_to_rreq_idx (batch_index);
 
    for (int j=0; j<steps_per_batch; j++) {
	for (int k=0; k<num_streams; k++) {
            MP_CHECK(mp_irecv ((void *)stream_state[k].buf_d, 
				size, peer*num_streams + k, &stream_state[k].reg, 
				&stream_state[k].rreq[req_idx + j]));
	}
    }
}

void wait_send (int batch_index) 
{
    int req_idx = batch_to_sreq_idx (batch_index); 

    for (int j=0; j<steps_per_batch; j++) {
        for (int k=0; k<num_streams; k++) {
            MP_CHECK(mp_wait(&stream_state[k].sreq[req_idx + j]));
        }
    }
}

void wait_recv (int batch_index) 
{
    int req_idx = batch_to_rreq_idx (batch_index);

    for (int j=0; j<steps_per_batch; j++) {
        for (int k=0; k<num_streams; k++) {
            MP_CHECK(mp_wait(&stream_state[k].rreq[req_idx + j]));
        }
    }
}

__global__ void send_op_kernel (mp::mlx5::send_desc_t desc)
{
    mp::device::mlx5::send(desc);
}

__global__ void wait_op_kernel (mp::mlx5::wait_desc_t desc)
{
    mp::device::mlx5::wait(desc);
    mp::device::mlx5::signal(desc);
}

__global__ void send_op_kernel_graph (mp::mlx5::send_desc_t *desc, unsigned int *index)
{
    unsigned int idx = *index;
    *index = (*index+1)%sindex_max_d;
    mp::device::mlx5::send(desc[idx]);
}

__global__ void wait_op_kernel_graph (mp::mlx5::wait_desc_t *desc, unsigned int *index)
{
    unsigned int idx = *index;
    *index = (*index+1)%windex_max_d;
    mp::device::mlx5::wait(desc[idx]);
    mp::device::mlx5::signal(desc[idx]);
}

void prepare_work_async_graphs (int size, int batch_index)
{
    int sreq_idx = batch_to_sreq_idx (batch_index);
    int rreq_idx = batch_to_rreq_idx (batch_index);

    for (int j=0; j<steps_per_batch; j++) {
        for(int k=0; k<num_streams; k++) {
            stream_state_t *curr_stream = (stream_state + k);   	
            int s_idx = sreq_idx + j;
            int r_idx = rreq_idx + j;

            if (!my_rank) {
                MP_CHECK(mp::mlx5::get_descriptors(&curr_stream->wdesc[r_idx], &curr_stream->rreq[r_idx]));

                MP_CHECK(mp_send_prepare((void *)curr_stream->buf_d, size, peer*num_streams + k, 
                            &curr_stream->reg, &curr_stream->sreq[s_idx]));
                MP_CHECK(mp::mlx5::get_descriptors(&curr_stream->sdesc[s_idx], 
                            &curr_stream->sreq[s_idx]));
            } else {
                MP_CHECK(mp_send_prepare((void *)curr_stream->buf_d, size, peer*num_streams + k, 
                            &curr_stream->reg, &curr_stream->sreq[s_idx]));
                MP_CHECK(mp::mlx5::get_descriptors(&curr_stream->sdesc[s_idx], &curr_stream->sreq[s_idx]));

                MP_CHECK(mp::mlx5::get_descriptors(&curr_stream->wdesc[r_idx], &curr_stream->rreq[r_idx]));
            }	
        }
    }
}


void graph_prepare_work (void *data)
{
    graph_preparation_arg_t *arg = (graph_preparation_arg_t *)data;
    prepare_work_async_graphs(arg->size, arg->batch_index);
}

void graph_post_work (void *data)
{
    graph_preparation_arg_t *arg = (graph_preparation_arg_t *)data;
    wait_recv(arg->batch_index);
    wait_send(arg->batch_index);
    ++arg->batch_index;
}

void capture_async_graph (int size, long long int kernel_size) 
{
    stream_state_t *curr_stream = stream_state;
    
    hipStreamBeginCapture(curr_stream->stream, hipStreamCaptureModeGlobal);

    CUDA_CHECK(hipLaunchHostFunc(curr_stream->stream, graph_prepare_work, graph_pre_arg));

    for (int j=0; j<steps_per_batch; j++) {
        if (!my_rank) {
            wait_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->wdesc_d, curr_stream->windex_d);
            CUDA_CHECK(hipGetLastError());

            if (kernel_size > 0) {
                if (use_calc_kernel > 0)
                    gpu_launch_calc_kernel(kernel_size, curr_stream->stream);
                else
                    poll_kernel <<<1, 1, 0, curr_stream->stream>>> (kernel_size);
            }

            CUDA_CHECK(hipGetLastError());

            send_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->sdesc_d, curr_stream->sindex_d);
            CUDA_CHECK(hipGetLastError());
        } else {
            send_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->sdesc_d, curr_stream->sindex_d);
            CUDA_CHECK(hipGetLastError());

            wait_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->wdesc_d, curr_stream->windex_d);
            CUDA_CHECK(hipGetLastError());

            if (kernel_size > 0) {
                if (use_calc_kernel > 0)
                    gpu_launch_calc_kernel(kernel_size, curr_stream->stream);
                else
                    poll_kernel <<<1, 1, 0, curr_stream->stream>>> (kernel_size);
            }
            CUDA_CHECK(hipGetLastError());
        }
    }

    CUDA_CHECK(hipLaunchHostFunc(curr_stream->stream, graph_post_work, graph_pre_arg));

    hipStreamEndCapture(curr_stream->stream, &graph);
    hipGraphInstantiate(&graphexec, graph, NULL, NULL, 0);
 
    hipStreamBeginCapture(curr_stream->stream, hipStreamCaptureModeGlobal);
    CUDA_CHECK(hipLaunchHostFunc(curr_stream->stream, graph_prepare_work, graph_pre_arg));
    for (int j=0; j<steps_per_batch; j++) {
        if (!my_rank) {
            wait_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->wdesc_d, curr_stream->windex_d);
            CUDA_CHECK(hipGetLastError());

            send_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->sdesc_d, curr_stream->sindex_d);
            CUDA_CHECK(hipGetLastError());
        } else {
            send_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->sdesc_d, curr_stream->sindex_d);
            CUDA_CHECK(hipGetLastError());

            wait_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->wdesc_d, curr_stream->windex_d);
            CUDA_CHECK(hipGetLastError());
        }
    }
    CUDA_CHECK(hipLaunchHostFunc(curr_stream->stream, graph_post_work, graph_pre_arg));
    hipStreamEndCapture(curr_stream->stream, &graph_comms);
    hipGraphInstantiate(&graphexec_comms, graph_comms, NULL, NULL, 0);
}

void create_async_graph (size_t size, long long int kernel_size) 
{
    std::vector<hipGraphNode_t> nodeDependencies, nodeDependencies2;
    hipGraphNode_t sendNode, waitNode, kernelNode, preNode, postNode;
    hipKernelNodeParams waitParams, sendParams, calcKernelParams, pollKernelParams;
    hipHostNodeParams preParams, postParams;
    hipGraphNode_t subgraphNode, subgraphNode_prev;

    CUDA_CHECK(hipGraphCreate(&graph, 0));
    CUDA_CHECK(hipGraphCreate(&graph_comms, 0));
    CUDA_CHECK(hipGraphCreate(&subgraph, 0));
    CUDA_CHECK(hipGraphCreate(&subgraph_comms, 0));

    preParams.fn = graph_prepare_work;
    preParams.userData = graph_pre_arg;

    postParams.fn = graph_post_work;
    postParams.userData = graph_pre_arg;

    for(int k=0; k<num_streams; k++) {
	stream_state_t *curr_stream = (stream_state + k); 

	waitParams.func = (void*)wait_op_kernel_graph;
        waitParams.gridDim = 1;
        waitParams.blockDim = 1;
        waitParams.sharedMemBytes = 0;
        void *waitArgs[2] = {(void*)&curr_stream->wdesc_d, (void *)&curr_stream->windex_d};
        waitParams.kernelParams = waitArgs;
        waitParams.extra = NULL;

        sendParams.func = (void*)send_op_kernel_graph;
        sendParams.gridDim = 1;
        sendParams.blockDim = 1;
        sendParams.sharedMemBytes = 0;
        void *sendArgs[2] = {(void*)&curr_stream->sdesc_d, (void *)&curr_stream->sindex_d};
        sendParams.kernelParams = sendArgs;
        sendParams.extra = NULL;

        const float value = 0.1F; 
        int n = kernel_size / sizeof(float);

        calcKernelParams.func = (void*)calc_kernel;
        calcKernelParams.gridDim = over_sub_factor * gpu_num_sm;
        calcKernelParams.blockDim = 32*2;
        calcKernelParams.sharedMemBytes = 0;
        void *calcKernelArgs[4] = {(void*)&n, (void *)&value, (void *)&curr_stream->in, (void *)&curr_stream->out};
        calcKernelParams.kernelParams = calcKernelArgs;
        calcKernelParams.extra = NULL;

        pollKernelParams.func = (void*)poll_kernel;
        pollKernelParams.gridDim = 1;
        pollKernelParams.blockDim = 1;
        pollKernelParams.sharedMemBytes = 0;
        void *pollKernelArgs[1] = {(void*)&kernel_size};
        pollKernelParams.kernelParams = pollKernelArgs;
        pollKernelParams.extra = NULL;

     	CUDA_CHECK(hipGraphCreate(&curr_stream->subgraph, 0));
        CUDA_CHECK(hipGraphCreate(&curr_stream->subgraph_comms, 0));

	//subgraph with comms + comp
        if (!my_rank) {
	   nodeDependencies.clear();
     	   CUDA_CHECK(hipGraphAddKernelNode(&waitNode, curr_stream->subgraph, nodeDependencies.data(), 
	        		   nodeDependencies.size(), &waitParams));

	   nodeDependencies.clear();
           nodeDependencies.push_back(waitNode);
           if (use_calc_kernel > 0) {
               CUDA_CHECK(hipGraphAddKernelNode(&kernelNode, curr_stream->subgraph, nodeDependencies.data(), 
	        		       nodeDependencies.size(), &calcKernelParams));
           } else { 
               CUDA_CHECK(hipGraphAddKernelNode(&kernelNode, curr_stream->subgraph, nodeDependencies.data(), 
	        		       nodeDependencies.size(), &pollKernelParams));
           }

	   nodeDependencies.clear();
           nodeDependencies.push_back(kernelNode);
           CUDA_CHECK(hipGraphAddKernelNode(&sendNode, curr_stream->subgraph, nodeDependencies.data(), 
			   	nodeDependencies.size(), &sendParams));
	} else {
	   nodeDependencies.clear();
           CUDA_CHECK(hipGraphAddKernelNode(&sendNode, curr_stream->subgraph, nodeDependencies.data(), 
	     		   	nodeDependencies.size(), &sendParams));

	   nodeDependencies.clear();
           nodeDependencies.push_back(sendNode);
     	   CUDA_CHECK(hipGraphAddKernelNode(&waitNode, curr_stream->subgraph, nodeDependencies.data(), 
	     		   nodeDependencies.size(), &waitParams));

	   nodeDependencies.clear();
           nodeDependencies.push_back(waitNode);
           if (use_calc_kernel > 0) {
               CUDA_CHECK(hipGraphAddKernelNode(&kernelNode, curr_stream->subgraph, nodeDependencies.data(), 
	     		       nodeDependencies.size(), &calcKernelParams));
           } else { 
               CUDA_CHECK(hipGraphAddKernelNode(&kernelNode, curr_stream->subgraph, nodeDependencies.data(), 
	     		       nodeDependencies.size(), &pollKernelParams));
           }
    	}

	//subgraph with comms
        if (!my_rank) {
	   nodeDependencies.clear();
     	   CUDA_CHECK(hipGraphAddKernelNode(&waitNode, curr_stream->subgraph_comms, nodeDependencies.data(), 
	        		   nodeDependencies.size(), &waitParams));

	   nodeDependencies.clear();
           nodeDependencies.push_back(waitNode);
           CUDA_CHECK(hipGraphAddKernelNode(&sendNode, curr_stream->subgraph_comms, nodeDependencies.data(), 
	     		   	nodeDependencies.size(), &sendParams));
	   
        } else {
	   nodeDependencies.clear();
           CUDA_CHECK(hipGraphAddKernelNode(&sendNode, curr_stream->subgraph_comms, nodeDependencies.data(), 
	     		   	nodeDependencies.size(), &sendParams));

	   nodeDependencies.clear();
           nodeDependencies.push_back(sendNode);
     	   CUDA_CHECK(hipGraphAddKernelNode(&waitNode, curr_stream->subgraph_comms, nodeDependencies.data(), 
	     		   nodeDependencies.size(), &waitParams));
           
    	}
    }

    //graph with compute and comms 
    CUDA_CHECK(hipGraphAddEmptyNode(&emptyNode, subgraph, NULL, 0));
    nodeDependencies.clear();
    nodeDependencies2.clear();
    nodeDependencies.push_back(emptyNode);
    for(int k=0; k<num_streams; k++) {
       CUDA_CHECK(hipGraphAddChildGraphNode(&subgraphNode, subgraph, nodeDependencies.data(), 
			       		nodeDependencies.size(), 
			       		stream_state[k].subgraph));
       nodeDependencies2.push_back(subgraphNode);
    }
    CUDA_CHECK(hipGraphAddEmptyNode(&emptyNode, subgraph, nodeDependencies2.data(), nodeDependencies2.size()));

    //graph with comms
    CUDA_CHECK(hipGraphAddEmptyNode(&emptyNode, subgraph_comms, NULL, 0));
    nodeDependencies.clear();
    nodeDependencies2.clear();
    nodeDependencies.push_back(emptyNode);
    for(int k=0; k<num_streams; k++) {
       CUDA_CHECK(hipGraphAddChildGraphNode(&subgraphNode, subgraph_comms, nodeDependencies.data(), nodeDependencies.size(), 
			       		stream_state[k].subgraph_comms));
       nodeDependencies2.push_back(subgraphNode);
    }
    CUDA_CHECK(hipGraphAddEmptyNode(&emptyNode, subgraph_comms, nodeDependencies2.data(), nodeDependencies2.size()));

    //create a graph for a batch of iterations
    //graph with compute and comms 
    CUDA_CHECK(hipGraphAddHostNode(&preNode, graph, NULL, 0, &preParams));
    nodeDependencies.clear();
    nodeDependencies.push_back(preNode);
    CUDA_CHECK(hipGraphAddChildGraphNode(&subgraphNode_prev, graph, nodeDependencies.data(), nodeDependencies.size(), subgraph_comms)); 
    for (int k=1; k<steps_per_batch; k++) {
        nodeDependencies.clear();
        nodeDependencies.push_back(subgraphNode_prev);
        CUDA_CHECK(hipGraphAddChildGraphNode (&subgraphNode, graph, nodeDependencies.data(), 
                    nodeDependencies.size(), subgraph));
        subgraphNode_prev = subgraphNode;
    }
    nodeDependencies.clear();
    nodeDependencies.push_back(subgraphNode_prev);
    CUDA_CHECK(hipGraphAddHostNode(&postNode, graph, nodeDependencies.data(), nodeDependencies.size(), &postParams));
    CUDA_CHECK(hipGraphInstantiate(&graphexec, graph, NULL, NULL, 0));

    //graph with comms 
    CUDA_CHECK(hipGraphAddHostNode(&preNode, graph_comms, NULL, 0, &preParams));
    nodeDependencies.clear();
    nodeDependencies.push_back(preNode);
    CUDA_CHECK(hipGraphAddChildGraphNode(&subgraphNode_prev, graph_comms, nodeDependencies.data(), nodeDependencies.size(), subgraph_comms)); 
    for (int k=1; k<steps_per_batch; k++) {
        nodeDependencies.clear();
        nodeDependencies.push_back(subgraphNode_prev);
        CUDA_CHECK(hipGraphAddChildGraphNode (&subgraphNode, graph_comms, nodeDependencies.data(), 
                    nodeDependencies.size(), subgraph_comms));
        subgraphNode_prev = subgraphNode;
    }
    nodeDependencies.clear();
    nodeDependencies.push_back(subgraphNode_prev);
    CUDA_CHECK(hipGraphAddHostNode(&postNode, graph_comms, nodeDependencies.data(), nodeDependencies.size(), &postParams));
    CUDA_CHECK(hipGraphInstantiate(&graphexec_comms, graph_comms, NULL, NULL, 0));
}

void destroy_async_graph () 
{
    CUDA_CHECK(hipGraphExecDestroy(graphexec));
    CUDA_CHECK(hipGraphExecDestroy(graphexec_comms));
    CUDA_CHECK(hipGraphDestroy(graph));
    CUDA_CHECK(hipGraphDestroy(graph_comms));
    if (!capture_graph) {
       CUDA_CHECK(hipGraphDestroy(subgraph));
       CUDA_CHECK(hipGraphDestroy(subgraph_comms));
       for(int k=0; k<num_streams; k++) {
           CUDA_CHECK(hipGraphDestroy(stream_state[k].subgraph));
           CUDA_CHECK(hipGraphDestroy(stream_state[k].subgraph_comms));
       }
    }
}

void trigger_work_async_kernels (int size, int batch_index, long long int kernel_size) 
{
    for (int j=0; j<steps_per_batch; j++) {
	CUDA_CHECK(hipEventRecord(iter_start_event, main_stream));	
	for(int k=0; k<num_streams; k++) {
	    stream_state_t *curr_stream = (stream_state + k);   	

   	    CUDA_CHECK(hipStreamWaitEvent(curr_stream->stream, iter_start_event, 0));	

	    if (!my_rank) {
	        wait_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->wdesc_d, curr_stream->windex_d);
                CUDA_CHECK(hipGetLastError());

                if (kernel_size > 0) {
                    if (use_calc_kernel > 0)
                       gpu_launch_calc_kernel(kernel_size, curr_stream->stream);
                    else
                       poll_kernel <<<1, 1, 0, curr_stream->stream>>> (kernel_size);
                }

	        send_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->sdesc_d, curr_stream->sindex_d);
                CUDA_CHECK(hipGetLastError());
 	    } else {
	       send_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->sdesc_d, curr_stream->sindex_d);
               CUDA_CHECK(hipGetLastError());

	       wait_op_kernel_graph<<<1,1,0,curr_stream->stream>>>(curr_stream->wdesc_d, curr_stream->windex_d);
               CUDA_CHECK(hipGetLastError());

               if (kernel_size > 0) {
                   if (use_calc_kernel > 0)
                      gpu_launch_calc_kernel(kernel_size, curr_stream->stream);
                   else
                      poll_kernel <<<1, 1, 0, curr_stream->stream>>>(kernel_size);
               }
 	    }

	    CUDA_CHECK(hipEventRecord(iter_stop_event, curr_stream->stream));
	    CUDA_CHECK(hipStreamWaitEvent(main_stream, iter_stop_event, 0));
	}
    }
}

void post_work_async_graphs (int size, int batch_index, long long int kernel_size)
{
    //prepare_work_async_graphs (size, batch_index);

    if (kernel_size) 
        CUDA_CHECK(hipGraphLaunch(graphexec, main_stream));
    else 
        CUDA_CHECK(hipGraphLaunch(graphexec_comms, main_stream));
}

void post_work_async_kernels (int size, int batch_index, long long int kernel_size) 
{
    for (int j=0; j<steps_per_batch; j++) {
	CUDA_CHECK(hipEventRecord(iter_start_event, main_stream));	  	

	for(int k=0; k<num_streams; k++) {
	    stream_state_t *curr_stream = (stream_state + k); 
            uint32_t r_idx = curr_stream->windex; 
            uint32_t s_idx = curr_stream->sindex; 
            curr_stream->windex = (curr_stream->windex+1)%windex_max;
            curr_stream->sindex = (curr_stream->sindex+1)%sindex_max;

      	    CUDA_CHECK(hipStreamWaitEvent(curr_stream->stream, iter_start_event, 0));	

	    if (!my_rank) {
                MP_CHECK(mp::mlx5::get_descriptors(&curr_stream->wdesc[r_idx], &curr_stream->rreq[r_idx]));
	        wait_op_kernel<<<1,1,0,curr_stream->stream>>>(curr_stream->wdesc[r_idx]);
                CUDA_CHECK(hipGetLastError());
	        
                if (kernel_size > 0) {
                    if (use_calc_kernel > 0)
                       gpu_launch_calc_kernel(kernel_size, curr_stream->stream);
                    else
                       poll_kernel <<<1, 1, 0, curr_stream->stream>>> (kernel_size);
                }

                MP_CHECK(mp_send_prepare((void *)stream_state[k].buf_d, size, peer*num_streams + k, 
	        			&curr_stream->reg, &curr_stream->sreq[s_idx]));
                MP_CHECK(mp::mlx5::get_descriptors(&curr_stream->sdesc[s_idx], &curr_stream->sreq[s_idx]));
	        send_op_kernel<<<1,1,0,curr_stream->stream>>>(curr_stream->sdesc[s_idx]);
                CUDA_CHECK(hipGetLastError());
 	    } else {
                MP_CHECK(mp_send_prepare((void *)stream_state[k].buf_d, size, peer*num_streams + k, 
	         		       &curr_stream->reg, &curr_stream->sreq[s_idx]));
                MP_CHECK(mp::mlx5::get_descriptors(&curr_stream->sdesc[s_idx], &curr_stream->sreq[s_idx]));
	        send_op_kernel<<<1,1,0,curr_stream->stream>>>(curr_stream->sdesc[s_idx]);
                CUDA_CHECK(hipGetLastError());

	        MP_CHECK(mp::mlx5::get_descriptors(&curr_stream->wdesc[r_idx], &curr_stream->rreq[r_idx]));
	        wait_op_kernel<<<1,1,0,curr_stream->stream>>>(curr_stream->wdesc[r_idx]);
                CUDA_CHECK(hipGetLastError());

                if (kernel_size > 0) {
                    if (use_calc_kernel > 0)
                       gpu_launch_calc_kernel(kernel_size, curr_stream->stream);
                    else
                       poll_kernel <<<1, 1, 0, curr_stream->stream>>> (kernel_size);
                }
 	    }

	    CUDA_CHECK(hipEventRecord(iter_stop_event, curr_stream->stream));
	    CUDA_CHECK(hipStreamWaitEvent(main_stream, iter_stop_event, 0));
	}
    }
}

void post_work_async (int size, int batch_index, long long int kernel_size) 
{
    int sreq_idx = batch_to_sreq_idx (batch_index);
    int rreq_idx = batch_to_rreq_idx (batch_index);
   
    for (int j=0; j<steps_per_batch; j++) {
	CUDA_CHECK(hipEventRecord(iter_start_event, main_stream));
	int s_idx = sreq_idx + j;
	int r_idx = rreq_idx + j;

	for(int k=0; k<num_streams; k++) {
	    stream_state_t *curr_stream = (stream_state + k); 
  	    CUDA_CHECK(hipStreamWaitEvent(curr_stream->stream, iter_start_event, 0));

	    if (!my_rank) { 
   	        MP_CHECK(mp_wait_on_stream(&curr_stream->rreq[r_idx], curr_stream->stream));

                if (kernel_size > 0) {
                    if (use_calc_kernel > 0)
                       gpu_launch_calc_kernel(kernel_size, curr_stream->stream);
                    else
                       poll_kernel <<<1, 1, 0, curr_stream->stream>>> (kernel_size);
                }

                MP_CHECK(mp_isend_on_stream ((void *)curr_stream->buf_d, size, peer*num_streams + k, 
	        			&curr_stream->reg, &curr_stream->sreq[s_idx], 
	        			curr_stream->stream));
   	    } else {
                MP_CHECK(mp_isend_on_stream ((void *)curr_stream->buf_d, size, peer*num_streams + k, 
	        			&curr_stream->reg, &curr_stream->sreq[s_idx], 
	        			curr_stream->stream));

                MP_CHECK(mp_wait_on_stream(&curr_stream->rreq[r_idx], curr_stream->stream));

                if (kernel_size > 0) {
                    if (use_calc_kernel > 0)
                       gpu_launch_calc_kernel(kernel_size, curr_stream->stream);
                    else
                       poll_kernel <<<1, 1, 0, curr_stream->stream>>> (kernel_size);
                }
	    }

	    CUDA_CHECK(hipEventRecord(iter_stop_event, curr_stream->stream));
	    CUDA_CHECK(hipStreamWaitEvent(main_stream, iter_stop_event, 0));
	}
   }
}

void post_work_sync (int size, int batch_index, long long int kernel_size) 
{
    int rreq_idx = batch_to_rreq_idx (batch_index);
    int sreq_idx = batch_to_sreq_idx (batch_index);

    for (int j=0; j<steps_per_batch; j++) {
	if (!my_rank) {
	    for(int k=0; k<num_streams; k++) {
	        stream_state_t *curr_stream = (stream_state + k); 

		MP_CHECK(mp_wait(&curr_stream->rreq[rreq_idx + j]));

                if (kernel_size > 0) {
                    if (use_calc_kernel > 0)
                       gpu_launch_calc_kernel(kernel_size, curr_stream->stream);
                    else
                       poll_kernel <<<1, 1, 0, curr_stream->stream>>> (kernel_size);
                }
	    }

	    for(int k=0; k<num_streams; k++) { 
	        stream_state_t *curr_stream = (stream_state + k); 

		CUDA_CHECK(hipStreamSynchronize(curr_stream->stream));

                MP_CHECK(mp_isend ((void *)curr_stream->buf_d, size, peer*num_streams + k, 
					&curr_stream->reg, &curr_stream->sreq[sreq_idx + j]));
	    }
       } else {
	    for(int k=0; k<num_streams; k++) { 
	        stream_state_t *curr_stream = (stream_state + k); 
                MP_CHECK(mp_isend ((void *)curr_stream->buf_d, size, peer*num_streams + k, 
					&curr_stream->reg, &curr_stream->sreq[sreq_idx + j]));
	    }

	    for(int k=0; k<num_streams; k++) { 
	        stream_state_t *curr_stream = (stream_state + k); 
                MP_CHECK(mp_wait(&curr_stream->rreq[rreq_idx + j]));

                if (kernel_size > 0) {
                    if (use_calc_kernel > 0)
                       gpu_launch_calc_kernel(kernel_size, curr_stream->stream);
                    else
                       poll_kernel <<<1, 1, 0, curr_stream->stream>>> (kernel_size);
                }
	    }

	    for(int k=0; k<num_streams; k++) { 
                CUDA_CHECK(hipStreamSynchronize(stream_state[k].stream));
	    }
        }
    }
}

double prepost_latency;

double sr_exchange (MPI_Comm comm, int size, int iter_count, int print_times, long long int kernel_size, 
            int use_async, int use_kernel_ops = 0, int use_graphs = 0)
{
    double time_start, time_stop;
    float cputime_elapsed, streamtime_elapsed;
    int batch_count, wait_send_batch = 0, wait_recv_batch = 0;
    int j;

    if (iter_count%steps_per_batch != 0) { 
	fprintf(stderr, "iter_count must be a multiple of steps_per_batch: %d \n", steps_per_batch);
	exit(-1);
    }
    batch_count = iter_count/steps_per_batch;
    tracking_event = 0;

    for (int i=0; i<num_streams; i++) { 
        stream_state_t *curr_stream = (stream_state + i); 
        CUDA_CHECK(hipMemset(curr_stream->buf_d, 0, size));
        curr_stream->sindex = curr_stream->windex = 0;
        CUDA_CHECK(hipMemset((void *)curr_stream->sindex_d, 
            		    0, sizeof(unsigned int)));
        CUDA_CHECK(hipMemset((void *)curr_stream->windex_d, 
        		    0, sizeof(unsigned int)));
    }
    CUDA_CHECK(hipDeviceSynchronize());

    post_recv (size, 0);

    MPI_Barrier(MPI_COMM_WORLD);

    *delay_flag = 0;
    if (use_async) { 
        CU_CHECK(hipStreamWaitValue32(main_stream, (hipDeviceptr_t)delay_flag_dptr, 1, hipStreamWaitValueEq));
        CUDA_CHECK(hipEventRecord(timer_start_event, main_stream));
    }
    time_start = MPI_Wtime();

    if (use_graphs) {
        graph_pre_arg->size = size;
        graph_pre_arg->batch_index = 0;
    }

    for (j=0; (j<batches_inflight) && (j<batch_count); j++) { 
        if (j<(batch_count-1)) {
            post_recv (size, j+1);
        }

        if (use_async) { 
            if (use_kernel_ops) {
                if (use_graphs) { 
                    post_work_async_graphs (size, j, kernel_size);
                } else { 
                    post_work_async_kernels (size, j, kernel_size);
                }
            } else { 
                post_work_async (size, j, kernel_size);
            }
        } else { 
            post_work_sync (size, j, kernel_size);
        }
    }

    *((volatile int *)delay_flag) = 1;

    wait_send_batch = wait_recv_batch = 0;
    while (wait_send_batch < batch_count) { 
        if (use_async) {
            if (!use_graphs)
                wait_recv (wait_recv_batch);
            wait_recv_batch++;
        }

        if (!use_graphs)
            wait_send (wait_send_batch);
        else
            CUDA_CHECK(hipStreamSynchronize(main_stream));
            
        wait_send_batch++;

        if (j < (batch_count-1)) {
            post_recv (size, j+1);
        }

        if (j < batch_count) { 
            if (use_async) { 
                if (use_kernel_ops) {
                    if (use_graphs) { 
                        post_work_async_graphs (size, j, kernel_size);
                    } else { 
                        post_work_async_kernels (size, j, kernel_size);
                    }
                } else { 
                    post_work_async (size, j, kernel_size);
                }
            } else { 
                post_work_sync (size, j, kernel_size);
            }

            j++;
        }
    }

    MPI_Barrier(comm);

    if (use_async) { 
        CUDA_CHECK(hipEventRecord(timer_stop_event, main_stream));
    }
    CUDA_CHECK(hipStreamSynchronize(main_stream));
    if (use_async) { 
        CUDA_CHECK(hipEventElapsedTime(&streamtime_elapsed, timer_start_event, timer_stop_event));
        streamtime_elapsed = ((streamtime_elapsed*1e3)/(iter_count*2)); 
    }
    time_stop = MPI_Wtime();
    cputime_elapsed = (time_stop - time_start);
    cputime_elapsed = ((cputime_elapsed*1e6)/((double)iter_count*2)); 

    if (print_times && !my_rank) {
        //if not using async, stream times do not make sense setting it from cputime
        if (!use_async) streamtime_elapsed = cputime_elapsed;
        fprintf(streamtimes, "%8.2lf \t", streamtime_elapsed);
        fprintf(cputimes, "%8.2lf \t", cputime_elapsed);
    }

    CUDA_CHECK(hipDeviceSynchronize());

    return (double)cputime_elapsed;
}

int main (int argc, char *argv[])
{
    int iter_count, max_size, size, dev_count, local_rank, dev_id = 0;
    long long int kernel_size = 20;
    int comm_comp_ratio = 1;
    int user_iter_count = 0;
    size = 1;
    max_size = MAX_SIZE;
    char line[200];
    char *value;

    value = getenv("ITER_COUNT");
    if (value != NULL) {
        user_iter_count = atoi(value);
    }

    value = getenv("ENABLE_DEBUG_MSG");
    if (value != NULL) {
        enable_debug_prints = atoi(value);
    }

    value = getenv("KERNEL_TIME");
    if (value != NULL) {
        kernel_size = atoi(value);
    }

    value = getenv("COMM_COMP_RATIO");
    if (value != NULL) {
        comm_comp_ratio = atoi(value);
    }

    value = getenv("CALC_SIZE");
    if (value != NULL) {
        calc_size = atoi(value);
    }

    use_calc_kernel = 0;
    value = getenv("USE_CALC_KERNEL");
    if (value != NULL) {
        use_calc_kernel = atoi(value);
    }

    value = getenv("STEPS_PER_BATCH");
    if (value != NULL) {
        steps_per_batch = atoi(value);
    }

    value = getenv("BATCHES_INFLIGHT");
    if (value != NULL) {
        batches_inflight = atoi(value);
    }

    value = getenv("SIZE");
    if (value != NULL && atoi(value)) {
        size = atoi(value);
    }

    value = getenv("NUM_STREAMS");
    if (value != NULL && atoi(value)) {
        num_streams = atoi(value);
    }

    value = getenv("USE_GRAPH_CAPTURE");
    if (value != NULL && atoi(value)) {
        capture_graph = atoi(value);
    }

    char tags[] = "wait_recv|wait_send|post_recv|post_work";

    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &comm_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &my_rank);

    if (comm_size != 2) { 
        fprintf(stderr, "this test requires exactly two processes \n");
        exit(-1);
    }

    CUDA_CHECK(hipGetDeviceCount(&dev_count));
    if (dev_count <= 0) {
        fprintf(stderr, "no CUDA devices found \n");
        exit(-1);
    }

    if (getenv("MV2_COMM_WORLD_LOCAL_RANK") != NULL) {
        local_rank = atoi(getenv("MV2_COMM_WORLD_LOCAL_RANK"));
    } else if (getenv("OMPI_COMM_WORLD_LOCAL_RANK") != NULL) {
        local_rank = atoi(getenv("OMPI_COMM_WORLD_LOCAL_RANK"));
    } else {
        local_rank = 0;
    }

    dev_id = local_rank;
    fprintf(stdout, "[%d] local_rank: %d dev_count: %d using GPU device: %d\n", my_rank, local_rank, dev_count, dev_id);

    CUDA_CHECK(hipSetDevice(dev_id));
    CUDA_CHECK(hipFree(0));

    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, dev_id));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(clockrate), (void *)&prop.clockRate, sizeof(int), 0, hipMemcpyHostToDevice));
    gpu_num_sm = prop.multiProcessorCount;

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(my_rank_d), (void *)&my_rank, sizeof(int), 0, hipMemcpyHostToDevice));

    fprintf(stdout, "[%d] GPU %d: %s PCIe %d:%d:%d\n", my_rank, dev_id, prop.name, prop.pciDomainID, prop.pciBusID, prop.pciDeviceID);

    peer = !my_rank;
    MP_CHECK(mp_init_multistream (MPI_COMM_WORLD, &peer, 1, MP_INIT_DEFAULT, dev_id, num_streams));

    iter_count = user_iter_count ? user_iter_count : ITER_COUNT_SMALL;
    if (!my_rank) { 
        cputimes = fopen("cputimes.txt", "w+");
        streamtimes = fopen("streamtimes.txt", "w+");
        if (!cputimes || !streamtimes)
        {
            printf("Could not open log file");
            exit(-1);
        }
        fprintf(stdout, "steps_per_batch: %d num_streams: %d batches_inflight: %d \n", 
                steps_per_batch, num_streams, batches_inflight);
        fprintf(stdout, "NOTE: printing half round-trip latency!!!\n");
    }

    stream_state = (stream_state_t *)malloc(sizeof(stream_state_t)*num_streams);
    NULL_CHECK(stream_state);

    graph_pre_arg = (graph_preparation_arg_t *)calloc(1, sizeof(graph_preparation_arg_t));
    NULL_CHECK(graph_pre_arg);

    sindex_max = batches_inflight*steps_per_batch;
    windex_max = (batches_inflight + 1)*steps_per_batch;
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(sindex_max_d), (void *)&sindex_max, sizeof(int), 
                0, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(windex_max_d), (void *)&windex_max, sizeof(int), 
                0, hipMemcpyHostToDevice));

    /*allocating requests*/
    for (int i=0; i<num_streams; i++) { 
        stream_state_t *curr = (stream_state + i); 
        int inflight_send_ops = steps_per_batch*batches_inflight;
        int inflight_recv_ops = steps_per_batch*(batches_inflight+1);
        curr->sreq = (mp_request_t *) malloc(inflight_send_ops*sizeof(mp_request_t));
        NULL_CHECK(curr->sreq);
        curr->rreq = (mp_request_t *) malloc(inflight_recv_ops*sizeof(mp_request_t));
        NULL_CHECK(curr->rreq);
        CUDA_CHECK(hipHostAlloc((void **)&curr->sdesc, inflight_send_ops*sizeof(mp::mlx5::send_desc_t), 0));
        CUDA_CHECK(hipHostAlloc((void **)&curr->wdesc, inflight_recv_ops*sizeof(mp::mlx5::wait_desc_t), 0));
        memset((void *)curr->sdesc, 0, inflight_send_ops*sizeof(mp::mlx5::send_desc_t));
        memset((void *)curr->wdesc, 0, inflight_recv_ops*sizeof(mp::mlx5::wait_desc_t));
        CUDA_CHECK(hipHostGetDevicePointer((void **)&curr->sdesc_d, curr->sdesc, 0));
        CUDA_CHECK(hipHostGetDevicePointer((void **)&curr->wdesc_d, curr->wdesc, 0));

        CUDA_CHECK(hipMalloc((void **)&curr->in, kernel_size));
        CUDA_CHECK(hipMalloc((void **)&curr->out, kernel_size));
        CUDA_CHECK(hipMalloc((void **)&curr->sindex_d, sizeof(unsigned int)));
        CUDA_CHECK(hipMalloc((void **)&curr->windex_d, sizeof(unsigned int)));
        CUDA_CHECK(hipMemset((void *)curr->in, 1, kernel_size));
        CUDA_CHECK(hipMemset((void *)curr->out, 1, kernel_size));

        CUDA_CHECK(hipStreamCreateWithFlags(&stream_state[i].stream, hipStreamNonBlocking));
    }

    CUDA_CHECK(hipStreamCreateWithFlags(&main_stream, hipStreamNonBlocking));
    CUDA_CHECK(hipHostAlloc(&delay_flag, sizeof(int), 0));
    CUDA_CHECK(hipHostGetDevicePointer((void **)&delay_flag_dptr, (void *)delay_flag, 0));
    CUDA_CHECK(hipEventCreateWithFlags(&iter_start_event, 0));
    CUDA_CHECK(hipEventCreateWithFlags(&iter_stop_event, 0));
    CUDA_CHECK(hipEventCreateWithFlags(&timer_start_event, 0));
    CUDA_CHECK(hipEventCreateWithFlags(&timer_stop_event, 0));

    if (!my_rank) {
        sprintf(line, "%10s \t %10s \t %10s \t %10s \t  %10s \t %10s \t %10s \t %10s \t %10s \t %10s  \n", 
                "MessageSize", "CompSize/Time", "CPU", "CPU+Comp", "MP", "MP+Comp", "MP-SM", 
                "MP-SM+Comp", "MP-Graph", "MP-Graph+Comp");
        fprintf(cputimes, "%s", line);
        fprintf(streamtimes, "%s", line);
    }

    if (size != 1) size = max_size = size;
    for (; size<=max_size; size*=2) {
        double latency;

        if (!my_rank) { 
            fprintf(stdout, "run for size: %10d in progress \n", size);
            fflush(stdout);
        }

        if (size > 1024) {
            iter_count = user_iter_count ? user_iter_count : ITER_COUNT_LARGE;
        }

        buf_size = size;

        for (int i=0; i<num_streams; i++) { 
            CUDA_CHECK(hipMalloc((void **)&stream_state[i].buf_d, buf_size));
            CUDA_CHECK(hipMemset(stream_state[i].buf_d, 0, buf_size)); 
            MP_CHECK(mp_register(stream_state[i].buf_d, buf_size, &stream_state[i].reg));
        }

        CUDA_CHECK(hipDeviceSynchronize());
        MPI_Barrier(MPI_COMM_WORLD);

        /*warmup base case*/
        sr_exchange(MPI_COMM_WORLD, size, iter_count, 0/*print times*/, 0, 0/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        /*calculate kenrel time based on base case latency*/
        latency = sr_exchange(MPI_COMM_WORLD, size, iter_count, 0/*print times*/, 0, 0/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        if (use_calc_kernel) 
            kernel_size = calc_size; 
        else  
            kernel_size = (comm_comp_ratio > 0) ? comm_comp_ratio*(latency) : kernel_size;

        if (!my_rank) {
            fprintf(cputimes, "%10d \t %10lld \t", size, kernel_size);
            fprintf(streamtimes, "%10d \t %10lld \t", size, kernel_size);
        }

        /*create graph*/
        if (capture_graph) { 
            if (num_streams != 1) {
                fprintf(stderr, "graph capture is only supported with single stream \n");
                exit (-1);
            }
            capture_async_graph (size, kernel_size); 
        } else { 
            create_async_graph (size, kernel_size);
        }

        hipProfilerStart();

        /*warmup for all variants*/
        sr_exchange(MPI_COMM_WORLD, size, iter_count, 0/*print times*/, kernel_size, 0/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        sr_exchange(MPI_COMM_WORLD, size, iter_count, 0/*print times*/, kernel_size, 1/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        sr_exchange(MPI_COMM_WORLD, size, iter_count, 0/*print times*/, kernel_size, 1/*use_async*/, 1/*use_kernel_ops*/);

        MPI_Barrier(MPI_COMM_WORLD);

        sr_exchange(MPI_COMM_WORLD, size, iter_count, 0/*print times*/, kernel_size, 1/*use_async*/, 1/*use_kernel_ops*/, 1/*use_graphs*/);

        MPI_Barrier(MPI_COMM_WORLD);

        /*Timed runs*/
        /*Normal*/
        sr_exchange(MPI_COMM_WORLD, size, iter_count, 1/*print times*/, 0 /*no kernel*/, 0/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        /*Normal + Kernel*/
        sr_exchange(MPI_COMM_WORLD, size, iter_count, 1/*print times*/, kernel_size, 0/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        /*Async*/
        sr_exchange(MPI_COMM_WORLD, size, iter_count, 1/*print times*/, 0/*kernel_size*/, 1/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        /*Async + kernel*/
        sr_exchange(MPI_COMM_WORLD, size, iter_count, 1/*print times*/, kernel_size, 1/*use_async*/);

        MPI_Barrier(MPI_COMM_WORLD);

        /*Async + Kernel Ops*/
        sr_exchange(MPI_COMM_WORLD, size, iter_count, 1/*print times*/, 0 /*kernel_size*/, 1/*use_async*/, 1/*use_kernel_ops*/);

        MPI_Barrier(MPI_COMM_WORLD);

        /*Async + Kernel + Kernel Ops*/
        sr_exchange(MPI_COMM_WORLD, size, iter_count, 1/*print times*/, kernel_size, 1/*use_async*/, 1/*use_kernel_ops*/);

        MPI_Barrier(MPI_COMM_WORLD);

        /*Async + Kernel Ops + Graphs*/
        sr_exchange(MPI_COMM_WORLD, size, iter_count, 1/*print times*/, 0 /*kernel_size*/, 1/*use_async*/, 1/*use_kernel_ops*/, 1/*use_graphs*/);

        MPI_Barrier(MPI_COMM_WORLD);

        /*Async + Kernel + Kernel Ops + Graphs*/
        sr_exchange(MPI_COMM_WORLD, size, iter_count, 1/*print times*/, kernel_size, 1/*use_async*/, 1/*use_kernel_ops*/, 1/*use_graphs*/);

        MPI_Barrier(MPI_COMM_WORLD);

        //hipProfilerStop();

        if (!my_rank) {
            fprintf(streamtimes, "\n");
            fprintf(cputimes, "\n");
        }

        /*destroy graphs*/
        destroy_async_graph();

        for (int i=0; i<num_streams; i++) {
            mp_deregister(&stream_state[i].reg);
            CUDA_CHECK(hipFree(stream_state[i].buf_d));
        }
    }

    if (!my_rank) {
        fprintf(cputimes, " \n");
        fprintf(streamtimes, " \n");

        rewind(cputimes);
        rewind(streamtimes);

        fprintf(stdout, "******************** CPU Timing ********************** \n");

        while(fgets(line, 200, cputimes) != NULL) 
            fprintf(stdout, "%s", line);
        fprintf(stdout, "\n");

        fprintf(stdout, "******************** STREAM Timing ********************** \n");

        while(fgets(line, 200, streamtimes) != NULL) 
            fprintf(stdout, "%s", line);
        fprintf(stdout, "\n");
    }

    for (int i=0; i<num_streams; i++) { 
        CUDA_CHECK(hipStreamDestroy(stream_state[i].stream));
        free(stream_state[i].sreq);
        free(stream_state[i].rreq);
    }
    CUDA_CHECK(hipStreamDestroy(main_stream));
    free(graph_pre_arg);
    free(stream_state);
    if (!my_rank) { 
        fclose(streamtimes);
        fclose(cputimes);
    }
    mp_finalize ();

    MPI_Barrier(MPI_COMM_WORLD);

    MPI_Finalize();

    return 0;
}
